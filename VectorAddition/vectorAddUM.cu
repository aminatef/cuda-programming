#include "hip/hip_runtime.h"
#include<stdio.h>
#include<cassert>
#include<iostream>

using std :: cout;

__global__ void vectorAddUm(int *a,int*b,int *c,int N){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id<N){
        c[id] = a[id]+b[id];
    }
}

int main(){
    int N = 50000;
    size_t size = sizeof(float) * N;
    int * a,*b,*c;
    hipMallocManaged(&a,size);
    hipMallocManaged(&b,size);
    hipMallocManaged(&c,size);


    // device id 

    int id  = hipGetDevice(&id);
    /*
    hipMallocManaged allows for oversubscription,
    and with the correct hipMemAdvise policies enabled,
    will allow the application to retain most if not all the performance of hipMalloc.
    hipMallocManaged also won't force an allocation to be resident until it is needed
    or prefetched, reducing the overall pressure on the operating system schedulers and
    better enabling multi-tenet use cases
    */
   hipMemAdvise(a,size,hipMemAdviseSetPreferredLocation,hipCpuDeviceId);
   hipMemAdvise(b,size,hipMemAdviseSetPreferredLocation,hipCpuDeviceId);

   hipMemPrefetchAsync(c,size,id);

   for(int i=0;i<N;i++){
    a[i] = rand()%100;
    b[i] = rand()%100;
   }
   hipMemAdvise(a,size,hipMemAdviseSetReadMostly,id);
   hipMemAdvise(b,size,hipMemAdviseSetReadMostly,id);
   hipMemPrefetchAsync(a,size,id);
   hipMemPrefetchAsync(b,size,id);
   int numThreads = 1024;
   int gridSize = (N + numThreads-1)/numThreads;
   vectorAddUm<<<gridSize , numThreads>>>(a,b,c,N);
   // We need this because we don't get the implicit synchronization of
   // hipMemcpy like in the original example
   hipDeviceSynchronize();
   hipMemPrefetchAsync(a,size,hipCpuDeviceId);
   hipMemPrefetchAsync(b,size,hipCpuDeviceId);
   hipMemPrefetchAsync(c,size,hipCpuDeviceId);

   for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }

  // Free unified memory (same as memory allocated with hipMalloc)
  hipFree(a);
  hipFree(b);
  hipFree(c);

  cout << "COMPLETED SUCCESSFULLY!\n";

  return 0;




}