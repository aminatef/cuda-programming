
#include <hip/hip_runtime.h>
#include<vector>
#include<algorithm>
#include <cstdlib>
#include<iostream>
#include<assert.h>
using std ::vector;
using std ::generate;
using std:: cout;
int RandomNumber(){
    return (rand()%100);
}

__global__ void matrixMul(const int * a ,const int *b ,int *c,int N){
    int raw = threadIdx.y + blockDim.y * blockIdx.y;
    int cal = threadIdx.x + blockDim.x * blockIdx.x;
    c[raw*N+cal] = 0;
    for (int i =0;i<N;i++){
        c[raw*N+cal] += a[raw*N+i]*b[cal+i*N];
    }
}

void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }
      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}


int main (){
    hipError_t error = hipSuccess;
    int numElements = 1024;
    size_t size =sizeof(int)*numElements*numElements;
    vector<int> h_a(numElements*numElements);
    vector<int> h_b(numElements*numElements);
    vector<int> h_c(numElements*numElements);

    generate(h_a.begin(),h_a.end(),RandomNumber);
    generate(h_b.begin(),h_b.end(),RandomNumber);


    // device memory allocation
    int *device_A = NULL;
    error = hipMalloc((void **)&device_A,size);
    if (error != hipSuccess){
        fprintf(stderr,"Falied to allocate device vector A\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    int *device_B = NULL;
    error = hipMalloc((void **)&device_B,size);
    if (error != hipSuccess){
        fprintf(stderr,"Falied to allocate device vector B\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    int *device_C = NULL;
    error = hipMalloc((void **)&device_C,size);
    if (error != hipSuccess){
        fprintf(stderr,"Falied to allocate device vector C\n",hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    error  = hipMemcpy(device_A,h_a.data(),size,hipMemcpyHostToDevice);
    error  = hipMemcpy(device_B,h_b.data(),size,hipMemcpyHostToDevice);
    int numThreads = 32;

    int numBlocks = (numElements+numThreads-1)/numThreads;
    dim3 threads(numThreads,numThreads);
    dim3 blocks(numBlocks,numBlocks);
    matrixMul<<<blocks,threads>>>(device_A,device_B,device_C,numElements);
    hipMemcpy(h_c.data(),device_C,size,hipMemcpyDeviceToHost);
    verify_result(h_a, h_b, h_c, numElements);

    cout << "COMPLETED SUCCESSFULLY\n";

    // Free memory on device
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

  return 0;


}