#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <cstdlib>
#include <iostream>
#include <assert.h>
#include"utils.h"
using std::cout;
using std::endl;
using std ::generate;
using std ::vector;

//OPTIMIZATION 2 
#define WPT 8 // WORK PER THREAD
#define TILE_SIZE  32 //TILE SIZE

//KERNEL CONFIG
const int numThreads = 32;
const int numElements = 1 << 7; 



__global__ void GEMM_opt2(int M, int N, int K, const float *A, const float *B, float *C)
{
    // all matrix are stored in col major format
    // more work per thread
    // A DIM -> k*M
    // B DIM -> N*k
    // C DIM -> M*N


    int global_row = threadIdx.x + blockIdx.x * TILE_SIZE;
    int global_col = threadIdx.y + blockIdx.y * TILE_SIZE;

    __shared__ float Asub[TILE_SIZE][TILE_SIZE];
    __shared__ float Bsub[TILE_SIZE][TILE_SIZE];

    int numTiles = K / TILE_SIZE;
    int RTS = TILE_SIZE / WPT;

    //INIT ACCUMALTE ARRAY
    float acc[WPT];
    for (int i = 0; i < WPT; i++)
    {
        acc[i] = 0.0f;
    }

    for (int i = 0; i < numTiles; i++)
    {
        for (int w = 0; w < WPT; w++)
        {
            int tiledRow = i * TILE_SIZE + threadIdx.x;
            int tiledCol = i * TILE_SIZE + threadIdx.y;
            Asub[threadIdx.y + w * RTS][threadIdx.x] = A[global_row + (tiledCol + w * RTS) * M];
            Bsub[threadIdx.y + w * RTS][threadIdx.x] = B[(global_col + w * RTS) * K + tiledRow];
        }
        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++)
        {
            for (int w = 0; w < WPT; w++)
            {
                acc[w] += Asub[j][threadIdx.x] * Bsub[threadIdx.y + w * RTS][j];
            }
        }
        __syncthreads();
        
    }

    for (int w = 0; w < WPT; w++)
    {
        C[(global_col + w * RTS) * M + global_row] = acc[w];
    }
}

void test_opt2(){
    size_t size = sizeof(float) * numElements * numElements;
    vector<float> h_a(numElements * numElements);
    vector<float> h_b(numElements * numElements);
    vector<float> h_c(numElements * numElements);

    generate(h_a.begin(), h_a.end(), RandomNumber);
    generate(h_b.begin(), h_b.end(), RandomNumber);

    // device memory allocation
    float *device_A = NULL;
    hipMalloc((void **)&device_A, size);
    float *device_B = NULL;
    hipMalloc((void **)&device_B, size);

    float *device_C = NULL;
    hipMalloc((void **)&device_C, size);

    hipMemcpy(device_A, h_a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(device_B, h_b.data(), size, hipMemcpyHostToDevice);

    int numBlocks = (numElements + numThreads - 1) / numThreads;
    dim3 threads(numThreads, numThreads/WPT);
    dim3 blocks(numBlocks, numBlocks);
    GEMM_opt2<<<blocks, threads>>>(numElements, numElements, numElements, device_B, device_A, device_C);
    hipMemcpy(h_c.data(), device_C, size, hipMemcpyDeviceToHost);
    verify_result(h_a, h_b, h_c,numElements);

    cout << "COMPLETED SUCCESSFULLY\n";

    // Free memory on device
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C); 
}

int main(){
    test_opt2();
}